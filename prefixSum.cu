#include <stdio.h>

#include <hip/hip_runtime.h>
#define min(a,b) ((a)<(b)?(a):(b))
#define MY_CUDA_CHECK(call) {                                    \
    hipError_t err = call;                                                    \
    if(hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    }}

#define DEFAULTBLOCKSIZE 1024
#define WARPSIZE 32
/*Warp prefix Sum*/
__device__ int ssb_warp_prefix_sum(int val,int*warpReduction) {
    int lane = threadIdx.x % WARPSIZE;
    int wid = threadIdx.x/WARPSIZE;

    int temp_val,old_val;
   /* UP_SWEEP */
    for (int offset = 2; offset <=WARPSIZE ; offset *= 2){
        temp_val=val;
        temp_val=((lane+1-offset/2)>=0)?__shfl_up(temp_val, offset/2):0;
        val += ((lane+1)%offset==0)?temp_val:0;
    }
    if((lane+1)==WARPSIZE){
        *warpReduction=val;
         val=0;
    }
    /* DOWN-SWEEP */
    for (int offset = WARPSIZE; offset >1 ; offset /= 2){// x[k+2​^(d+1)​–1]=x[k+2^​d​–1]+x[k+ 2​^(d+​1)–1] 

        temp_val=val;
        old_val=val;

        __syncthreads();

        old_val=__shfl_down(old_val, offset/2);
        val=  ( ((lane+1)%(offset/2)==0) && ((lane+1)%offset !=0) )?old_val:val;

        temp_val=((lane+1-offset/2)>=0)?__shfl_up(temp_val, offset/2):0;
        val += ((lane+1)%offset==0)?temp_val:0;
        
    }

	return val;
}

__device__ int warp_reduction(int val){
    int lane=threadIdx.x%WARPSIZE;
    int temp;
    for(int off=1; off<=WARPSIZE/2;off*=2){
        temp=__shfl_up(val,off);
        val+=(lane>=off)?temp:0;
    }

    return val;
}
__global__ void ssb_prefix_sum(int* in,int* out,int N){
    int t_x= threadIdx.x;
    int wid = threadIdx.x/WARPSIZE;
    int lane=t_x%WARPSIZE;
    int j=0;
    int val,valIn,lastLane;
    int shared_dim=DEFAULTBLOCKSIZE/WARPSIZE;//(DEFAULTBLOCKSIZE +WARPSIZE-1)/WARPSIZE;
    static __shared__ int block_prefix[DEFAULTBLOCKSIZE/WARPSIZE];
    static __shared__ int first;

    if(t_x==0){
        first=0;
    }
    do{ 
        lastLane= min(DEFAULTBLOCKSIZE,N-j);
        valIn=in[t_x+j];//((t_x+j)<N)?in[t_x+j]:0;
        /*  in block prefix[i] sono presenti la somma di tutti gli elementi del warp i
            questo valore è ricavato dall'ultimo thread del warp nella fase di prefix Sum   */
        val = ssb_warp_prefix_sum(valIn,&block_prefix[wid]);   

        __syncthreads();
        /*  effettuando una riduzione in un warp troviamo in posizione [i] la somma di tutti gli elementi
            del warp [i] e di quelli prima        */
        if(wid==0){
            int temporal= (lane< shared_dim)?block_prefix[lane]:0;
            temporal= warp_reduction(temporal);
            if(lane< shared_dim)
                block_prefix[lane] = temporal ;
        }
        /* si scrive in memoria globale tenendo conto dell'offset cumulato e l'ultimo thread 
           del blocco con indice valido aggiorna il valore di first utile al prossimo turno */
        __syncthreads();
        if(t_x < lastLane)
            out[t_x+j]=(wid==0)?val+first:val+block_prefix[wid-1]+first;
        //__syncthreads();
        if(t_x == (lastLane - 1)){
            first+= block_prefix[wid];
        }
        __syncthreads();
        j+=blockDim.x;

    }while(j<N);

}



// This function verifies the shuffle scan result, for the simple
// prefix sum case.
bool CPUverify(int *h_data, int *h_result, int n_elements)
{
    // cpu verify
    for (int i=0; i<n_elements-1; i++)
    {
        h_data[i+1] = h_data[i] + h_data[i+1];
    }

    int diff = 0;

    for (int i=0 ; i<n_elements; i++)
    {
    	//printf("%d\n",h_data[i]);
        diff += h_data[i]-h_result[i];
        //nella versione cpu aggiunge anche se stesso(prefix[x]+x)
    }
    diff-=h_data[n_elements-1];
    printf("CPU verify result diff (GPUvsCPU) = %d\n", diff);
    bool bTestResult = false;

    if (diff == 0) bTestResult = true;

    return bTestResult;
}

int main(int argc, char **argv) {
    int *h_data, *h_result;
    int *d_data,*d_out;
    int blockSize = DEFAULTBLOCKSIZE;
    int n_elements= 65536;
    int n_aligned;
    if(argc>1) {
    	n_elements = atoi(argv[1]);
    }
    n_aligned=((n_elements+blockSize-1)/blockSize)*blockSize;
    int sz = sizeof(int)*n_aligned;

    printf("Starting shfl_scan\n");

    MY_CUDA_CHECK(hipHostMalloc((void **)&h_data, sizeof(int)*n_aligned, hipHostMallocDefault));
    MY_CUDA_CHECK(hipHostMalloc((void **)&h_result, sizeof(int)*n_elements, hipHostMallocDefault));

    //initialize data:
    printf("Computing Simple Sum test on %d (%d) elements\n",n_elements, n_aligned);
    printf("---------------------------------------------------\n");

    printf("Initialize test data\n");
    //char line[1024];
    for (int i=0; i<n_elements; i++)
    {
        h_data[i] = i;
//        fgets(line,sizeof(line),stdin);
//        sscanf(line,"%d",&h_data[i]);
    }

    for (int i=n_elements; i<n_aligned; i++) {
	h_data[i] = 0;
    }

    printf("Scan summation for %d elements\n", n_elements);

    // initialize a timer
    hipEvent_t start, stop;
    MY_CUDA_CHECK(hipEventCreate(&start));
    MY_CUDA_CHECK(hipEventCreate(&stop));
    float et = 0;
    float inc = 0;

    MY_CUDA_CHECK(hipMalloc((void **)&d_data, sz));
    MY_CUDA_CHECK(hipMalloc((void **)&d_out, sz));

    MY_CUDA_CHECK(hipMemcpy(d_data, h_data, sz, hipMemcpyHostToDevice));
    //ssb_prefix_sum<<< 1,blockSize>>>(d_data,d_out,n_elements);
    ssb_prefix_sum<<< 1,blockSize>>>(d_data,d_out,n_aligned);
    MY_CUDA_CHECK(hipEventRecord(start, 0));
    MY_CUDA_CHECK(hipEventRecord(stop, 0));
    MY_CUDA_CHECK(hipEventSynchronize(stop));
    MY_CUDA_CHECK(hipEventElapsedTime(&inc, start, stop));
    et+=inc;
    MY_CUDA_CHECK(hipMemcpy(h_result, d_out, n_elements*sizeof(int), hipMemcpyDeviceToHost));
    printf("\n");
   /*  for(int i =0;i< n_elements;i++){
        if(i%WARPSIZE==0)
            printf("\n%d) %d\n",i/WARPSIZE,h_data[i]);
        printf("%d ",h_result[i]);
        
    }
     */
//    MY_CUDA_CHECK(cudaMemcpy(h_result, d_data, n_elements*sizeof(int), cudaMemcpyDeviceToHost));
    printf("\nTime (ms): %f\n", et);
    printf("%d elements scanned in %f ms -> %f MegaElements/s\n",
             n_elements, et, n_elements/(et/1000.0f)/1000000.0f);

    bool bTestResult = CPUverify(h_data, h_result, n_elements);

    MY_CUDA_CHECK(hipHostFree(h_data));
    MY_CUDA_CHECK(hipHostFree(h_result));

    return (int)bTestResult;
}