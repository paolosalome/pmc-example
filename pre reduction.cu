
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>

#define min(a,b) (a) < (b) ? (a) : (b)
#define BOOL int
#define blockSide 16
#define blockNum 16
#define epsilon 1e-4
#define N 28
#define M 56
#define P 20000
#define DATA float
#define eta 0.05f
#define alpha 0.8f


static void HandleCuda(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_CUDA( err ) (HandleCuda( err, __FILE__, __LINE__ ))

void startTimer(hipEvent_t *start, hipEvent_t *stop) {
	HANDLE_CUDA(hipEventCreate(start));
	HANDLE_CUDA(hipEventCreate(stop));
	HANDLE_CUDA(hipEventRecord(*start, 0));
}

void stopAndPrint(hipEvent_t *start, hipEvent_t *stop) {
	HANDLE_CUDA(hipEventRecord(*stop, 0));
	HANDLE_CUDA(hipEventSynchronize(*stop));
	float time = 0.0f;
	HANDLE_CUDA(hipEventElapsedTime(&time, *start, *stop));
	printf("Elapsed Time: %f milliseconds\n", time);
	HANDLE_CUDA(hipEventDestroy(*start));
	HANDLE_CUDA(hipEventDestroy(*stop));
}


/* la matrice di destinazione è width_h2h x width_delta     */
/* h2h_corner,delta_corner  sono in previsione di una "sliding grid" */
__device__ void matrix_array_block(DATA* h2h, DATA* w,  DATA* delta_weight, DATA* delta_bias, DATA* delta, DATA* thr_delta_W, DATA* dest_delta, DATA* delta_weight_dest, DATA* delta_bias_dest, int width_h2h, int width_delta, int A_right_limit, int B_right_limit,int stream,BOOL enable_bias){
    int t_x = threadIdx.x;
    int t_y = threadIdx.y;
    int idx = t_x + blockIdx.x*blockSide ; 
    int idy = t_y + blockIdx.y*blockSide ;
    int pattern;
    int h2h_corner = blockIdx.y*blockSide;
    int delta_corner = blockIdx.x*blockSide;
    
    __shared__ DATA temp_shifted_mul[blockSide][blockSide*blockSide];//può contenere diversi 0 nei casi sui bordi
    __shared__ DATA temp_sum_delta_h2h[blockSide*blockSide];//può essere riciclato per W
    __shared__ DATA block_h2h[blockSide*blockSide];
    __shared__ DATA block_w[blockSide*(blockSide+1)];//usefull for avoid bank conflict
    __shared__ DATA block_delta[blockSide*blockSide];
    __shared__ DATA bias_to_update[blockSide*blockSide];

    int max_b_x = ((delta_corner + blockSide) < B_right_limit) ? blockSide: (B_right_limit - delta_corner);
    int max_a_x = ((h2h_corner + blockSide) < A_right_limit) ? blockSide: (A_right_limit - h2h_corner);

    temp_sum_delta_h2h[t_x+t_y*blockSide]=0.0f;
    block_w[t_x*blockSide+t_y] = (max_a_x > t_y && max_b_x > t_x) ? w[t_y*width_delta + t_x]:0.0f;
    bias_to_update[t_y*blockSide + t_x] = 0.0f;//block_delta[t_x*blockSide+t_y]; si considera la trasposta per fare la riduzione sul blocco

    for(int curr_patterns=0;curr_patterns<P;curr_patterns+=blockSide){
        pattern = (curr_patterns  + blockSide > P) ? (P-curr_patterns): blockSide ;

        //DATA val = ((curr_patterns+ t_y) < P && max_b_x > t_x) ? delta[t_y*width_delta + t_x + curr_patterns*width_delta]:0.0f;

        block_h2h[t_y*blockSide+t_x]= ((curr_patterns+ t_y) < P && max_a_x > t_x) ? h2h[t_y*width_h2h + t_x +curr_patterns*width_h2h]:0.0f;
        block_delta[t_y*blockSide+t_x] = ((curr_patterns+ t_y) < P && max_b_x > t_x) ? delta[t_y*width_delta + t_x + curr_patterns*width_delta]:0.0f;
        __syncthreads();

        DATA val = block_delta[t_y*blockSide+t_x];
        DATA temp=0.0f;
        
        for(int i=0 ;i<blockSide;i++){
        /*  QUI CI VA IL PRODOTTO TRA TEMP=W*DELTA   */
            temp += block_delta[t_y*blockSide+i]*block_w[i*blockSide+t_x];//product delta*W by trd[ty][tx]
            temp_shifted_mul[t_y][ t_x + i*blockSide ] =  val*block_h2h[t_y*blockSide+i];
           //temp_shifted_mul[t_y][ t_x + i*blockSide ] =  ( (curr_patterns+ t_y) < P && max_a_x > i) ? val*h2h[t_y*width_h2h + i +curr_patterns*width_h2h]:0.0f;
        }    

        __syncthreads();
        atomicAdd(&dest_delta[t_y*width_h2h+ curr_patterns*width_h2h + t_x], temp*block_h2h[t_y*blockSide+t_x]*(1-block_h2h[t_y*blockSide+t_x]));//product 
        if(t_y==0){
            for(int j=t_x,index=0; index<blockSide;j+=blockSide, index++ ){
                for(int i=0 ;i<pattern;i++){
                    temp_sum_delta_h2h[j] += eta*temp_shifted_mul[i][j];
                }
            }
        }
        __syncthreads();
        if(enable_bias==1)//solo i blocchi con blocky = 0
            bias_to_update[t_y*blockSide + t_x] += block_delta[t_y*blockSide + t_x];
    }
    if(t_y + h2h_corner < A_right_limit && t_x + delta_corner < B_right_limit)
        thr_delta_W[t_x+t_y*width_delta] = temp_sum_delta_h2h[t_y*blockSide+ t_x];// alpha*delta_weight[t_x+t_y*width_delta]:0.0f;  
        /*+ alpha * deltaWeight[t_x+t_y*width_delta] da mettere nella riduzione delle matrici costruite tra i vari stream*/
    if(enable_bias==1 &&  t_y==0){
        DATA temp=0.0f;
        for(int i=0;i<blockSide;i++)
            temp+=bias_to_update[i*blockSide+t_x];
        delta_bias_dest[t_x] = eta*temp + alpha*delta_bias[t_x];
       // printf("enable_bias [%d][%d] %d temp:%f\n",idy,idx,enable_bias,temp);
    }
}
/* IMPORTANTE : MANCA RIDUZIONE DI  DELTA W-H2H volendo si incorpora anche la riduzione di bias */
/* si può la riduzione finale di W sommando i delta calcolati e riaggiornare quindi il delta W con gli stessi .
 oppure si fa prima ma bisogna salvarlo a parte e non sovrascrivere subito la matrice di partenza (problemi di concorrenza con altri stream) 
 la matrice di desinazione avrà streams*L*(L+1) elementi . si effettua la riduzione sullo stream principale per salvarla su quella giusta
 */


/* THR_DEST has nupl[L]*nupl[l+1] element*/

__global__ void matrix_mul(DATA* H2H, DATA* W, DATA* DELTA_WEIGHT, DATA* DELTA_BIAS, DATA* DELTA, DATA* THR_DELTA_W_H2H, DATA* DEST_DELTA,DATA* DELTA_WEIGHT_DEST, DATA* DELTA_BIAS_DEST, int width_h2h, int width_delta, int A_right_limit, int B_right_limit, int stream, BOOL enable_bias){
    int b_x = blockIdx.x*blockSide;
    int b_y = blockIdx.y*blockSide;
    //enable bias vale 1 se la griglia si è spostata lungo la x . Gli unici blocchi che calcoleranno il delta bias sono quelli con blockIdy = 0
    if(b_x < B_right_limit && b_y <A_right_limit)
        matrix_array_block(H2H +b_y, W +b_x+b_y*width_delta, DELTA_WEIGHT +b_x+b_y*width_delta, DELTA_BIAS +b_x, DELTA +b_x, THR_DELTA_W_H2H +b_x+b_y*width_delta, DEST_DELTA+b_y, DELTA_WEIGHT_DEST +b_x+b_y*width_delta, DELTA_BIAS_DEST +b_x, width_h2h, width_delta, A_right_limit, B_right_limit, stream, enable_bias*(1-blockIdx.y));
    __syncthreads();
}
void optimum_grid_x(dim3* grid,int max_block,int y_limit){
    
    int x = min((N+blockSide-1)/blockSide,max_block);
    int y=max_block/x;
    int prod=x*y;
    int new_prod;
    int new_x;

    for(new_x=x, new_prod=new_x*y ; new_prod != max_block && new_x > 1 && y < y_limit ;new_x--){
        y=max_block/new_x;
        new_prod=new_x*y;
    }

    if(new_prod>prod)
        x= new_x+1;

    grid->x = x;
    grid->y = y;

}                                                                    
void backward(DATA *host_h2h, DATA* host_delta, DATA* host_thread_delta, DATA* d_h2h, DATA* d_w, DATA* d_delta_weight, DATA* d_delta_bias, DATA* d_delta, DATA* d_thread_delta, DATA* d_dest_delta, DATA* d_delta_weight_dest, DATA* d_delta_bias_dest, int width_h2h, int width_delta){
    dim3 grid,block;
    optimum_grid_x(&grid,blockNum,width_h2h/blockSide);
    block.x= blockSide;
    block.y= blockSide;
    printf("grid :%d %d\n",grid.y,grid.x);
    hipEvent_t start,stop;
    startTimer(&start,&stop);
    for(int sw_x=0; sw_x < width_delta; sw_x += grid.x*blockSide)
        for(int sw_y=0; sw_y < width_h2h;sw_y += grid.y*blockSide) {                                            
            matrix_mul<<< grid,block >>>(d_h2h +sw_y, d_w +sw_x+sw_y*width_delta, d_delta_weight +sw_x+sw_y*width_delta, d_delta_bias +sw_x, d_delta +sw_x, d_thread_delta+sw_x+sw_y*width_delta, d_dest_delta + sw_y, d_delta_weight_dest +sw_x+sw_y*width_delta, d_delta_bias_dest +sw_x, width_h2h, width_delta, min(width_h2h-sw_y,grid.y*blockSide) ,min(width_delta-sw_x,grid.x*blockSide),0,1-sw_y);
        }
    stopAndPrint(&start,&stop);

}

/*Check device*/
BOOL matsAreEquals(DATA *A, DATA *B, int rows, int cols) {

	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) { // the first column is for adapting the data
			float err = fabs(A[i*cols + j] - B[i*cols + j]);
			//printf("Error in i=%d,j=%d: %f\n", i, j, err);
			if (err >= epsilon) { printf("row: %d, col: %d\n", i, j); return 0; }
		}
	}
	return 1;
}

void printMat(DATA *mat, int rows, int cols) {

	for (int i = 0; i < rows; i++) {
		printf("ROW %d : {", i);
		for (int j = 0; j < cols; j++) {
			printf("%f - ", mat[i*cols + j]);
		}
		printf("}");
		printf("\n\n");
	}
	printf("\n\n");
}

int main(){
    DATA *h2h, *w, *bias, *delta, *c_host, *dest_c, *new_delta, *delta_host, *delta_weight, *new_delta_weight, *delta_bias, *new_delta_bias;
    DATA *d_h2h, *d_w, *d_bias,*d_delta, *d_thread_delta, *d_dest_delta, *d_delta_weight, *d_delta_bias, *d_delta_weight_dest, *d_delta_bias_dest;

    h2h=(DATA *)malloc(P*M*sizeof(DATA));
    w=(DATA *)malloc(M*N*sizeof(DATA));
    bias=(DATA *)malloc(M*sizeof(DATA));
    delta=(DATA *)malloc(P*N*sizeof(DATA));//delta h2h
    new_delta=(DATA *)calloc(P*M,sizeof(DATA));
    delta_host=(DATA *)calloc(P*M,sizeof(DATA));
    c_host=(DATA *)calloc(M*N,sizeof(DATA));
    dest_c=(DATA *)calloc(M*N,sizeof(DATA));
    new_delta_weight=(DATA *)calloc(M*N,sizeof(DATA));
    delta_weight=(DATA *)calloc(M*N,sizeof(DATA));
    new_delta_bias=(DATA *)calloc(N,sizeof(DATA));
    delta_bias=(DATA *)calloc(N,sizeof(DATA));

    hipMalloc((void**)&d_h2h,P*M*sizeof(DATA));
    hipMalloc((void**)&d_w,M*N*sizeof(DATA));
    hipMalloc((void**)&d_bias,N*sizeof(DATA));
    hipMalloc((void**)&d_delta,P*N*sizeof(DATA));
    hipMalloc((void**)&d_dest_delta,P*M*sizeof(DATA));
    hipMalloc((void**)&d_thread_delta,M*N*sizeof(DATA));
    hipMalloc((void**)&d_delta_weight,M*N*sizeof(DATA));
    hipMalloc((void**)&d_delta_bias,N*sizeof(DATA));
    hipMalloc((void**)&d_delta_weight_dest,M*N*sizeof(DATA));
    hipMalloc((void**)&d_delta_bias_dest,N*sizeof(DATA));

/* -------------------------------init  -------------------*/
    for(int row=0;row<P;row++){
        for(int cola=0;cola<M;cola++)
            h2h[row*M+cola]=(DATA)rand() / (DATA)RAND_MAX;
        for(int colb=0;colb<N;colb++)
            delta[row*N+colb]=(DATA)rand() / (DATA)RAND_MAX;      
    }
    for(int colb=0;colb<N;colb++){
        //bias[colb]=(DATA)rand() / (DATA)RAND_MAX;
        delta_bias[colb]=(DATA)rand() / (DATA)RAND_MAX;
        for(int cola=0;cola<M;cola++){
            w[cola*N+colb]=(DATA)rand() / (DATA)RAND_MAX;
            delta_weight[cola*N+colb]=(DATA)rand() / (DATA)RAND_MAX;
        }
    }
/*  -------------------------------------   */
    hipMemcpy(d_h2h,h2h,P*M*sizeof(DATA),hipMemcpyHostToDevice);
    hipMemcpy(d_w,w,M*N*sizeof(DATA),hipMemcpyHostToDevice);
    //cudaMemcpy(d_bias,bias,N*sizeof(DATA),cudaMemcpyHostToDevice);
    hipMemcpy(d_delta,delta,P*N*sizeof(DATA),hipMemcpyHostToDevice);
    hipMemcpy(d_dest_delta,new_delta,P*M*sizeof(DATA),hipMemcpyHostToDevice);
    hipMemcpy(d_thread_delta,c_host,M*N*sizeof(DATA),hipMemcpyHostToDevice);//parte di delta_weight nuovo
    hipMemcpy(d_delta_weight,delta_weight,M*N*sizeof(DATA),hipMemcpyHostToDevice);
    hipMemcpy(d_delta_bias,delta_bias,N*sizeof(DATA),hipMemcpyHostToDevice);
    hipMemcpy(d_delta_weight_dest,new_delta_weight,M*N*sizeof(DATA),hipMemcpyHostToDevice);
    hipMemcpy(d_delta_bias_dest,new_delta_bias,N*sizeof(DATA),hipMemcpyHostToDevice);

    backward(h2h, delta, dest_c, d_h2h, d_w, d_delta_weight, d_delta_bias, d_delta, d_thread_delta, d_dest_delta, d_delta_weight_dest, d_delta_bias_dest, M, N);
    
    for(int row=0;row<P;row++){
        for(int cola=0;cola<M;cola++){
            DATA temp= 0.0f;
            for(int colb=0;colb<N;colb++)
                temp+= delta[row*N+colb]*w[cola*N+colb];    
            delta_host[row*M+cola] = temp*h2h[row*M+cola]*(1-h2h[row*M+cola]);
        }
    }
    for(int colb=0;colb<N;colb++)
        new_delta_bias[colb] = alpha*delta_bias[colb];
    for(int row=0;row<P;row++){
        for(int colb=0;colb<N;colb++){
            new_delta_bias[colb] += eta*delta[row*N+colb] ;
            for(int cola=0;cola<M;cola++)
                c_host[cola*N+colb]+= eta*h2h[row*M+cola]* delta[row*N+colb];// + (row==0)?alpha * delta_weight[cola][colb]:0.0f;
        }
    }

    hipMemcpy(dest_c,d_thread_delta, M*N*sizeof(DATA),hipMemcpyDeviceToHost);
    hipMemcpy(new_delta,d_dest_delta, P*M*sizeof(DATA),hipMemcpyDeviceToHost);
    hipMemcpy(delta_bias,d_delta_bias_dest, N*sizeof(DATA),hipMemcpyDeviceToHost);
    hipMemcpy(delta_weight,d_delta_weight_dest, M*N*sizeof(DATA),hipMemcpyDeviceToHost);

    /*printMat(c_host,M,N);
    printf("------------------------------\n");
    printMat(dest_c,M,N);
    printf("------------------------------\n");*/
    matsAreEquals(dest_c,c_host,M,N);
    printf(" delta h2h +++\n");
    matsAreEquals(new_delta,delta_host,P,M);
    printf(" delta W-h2h+++\n");
    matsAreEquals(new_delta_bias,delta_bias,1,N);
    printMat(new_delta_bias,1,N);
    printMat(delta_bias,1,N);

    
        

    //printMat(new_delta,P,M);

    free(h2h);
    free(w);
    free(delta);
    free(delta_host);
    free(c_host);
    free(dest_c);
    hipFree(d_h2h);
    hipFree(d_w);
    hipFree(d_delta);
    hipFree(d_thread_delta);
    hipFree(d_dest_delta);
    return 0;
}