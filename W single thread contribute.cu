
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>

#define min(a,b) (a) < (b) ? (a) : (b)
#define BOOL int
#define blockSide 16
#define blockNum 16
#define epsilon 1e-5
#define N 58
#define M 784
#define P 20000
#define DATA float


static void HandleCuda(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_CUDA( err ) (HandleCuda( err, __FILE__, __LINE__ ))

void startTimer(hipEvent_t *start, hipEvent_t *stop) {
	HANDLE_CUDA(hipEventCreate(start));
	HANDLE_CUDA(hipEventCreate(stop));
	HANDLE_CUDA(hipEventRecord(*start, 0));
}

void stopAndPrint(hipEvent_t *start, hipEvent_t *stop) {
	HANDLE_CUDA(hipEventRecord(*stop, 0));
	HANDLE_CUDA(hipEventSynchronize(*stop));
	float time = 0.0f;
	HANDLE_CUDA(hipEventElapsedTime(&time, *start, *stop));
	printf("Elapsed Time: %f milliseconds\n", time);
	HANDLE_CUDA(hipEventDestroy(*start));
	HANDLE_CUDA(hipEventDestroy(*stop));
}

/* __inline__ __device__ DATA warpGetVal(DATA val, int offset) {
    return __shfl(val, offset);
}
 */
/* la matrice di destinazione è col1 x col2     */
/* a_corner,b_corner  sono in previsione di una "sliding grid" */

__device__ void matrix_array_block(DATA* h2h, DATA* w, DATA* delta, DATA* thr_delta, DATA* dest_delta, int col1, int col2, int A_right_limit, int B_right_limit){
    int t_x = threadIdx.x;
    int t_y = threadIdx.y;
 /*    int block_x = blockIdx.x*blockSide;
    int block_y = blockIdx.y*blockSide;
    int idx = t_x + block_x ; 
    int idy = t_y + block_y ; */
    int pattern;
    int a_corner = blockIdx.y*blockSide;
    int b_corner = blockIdx.x*blockSide;

    __shared__ DATA temp_shifted_mul[blockSide][blockSide*blockSide];//può contenere diversi 0 nei casi sui bordi
    __shared__ DATA temp_sum_delta_h2h[blockSide*blockSide];//può essere riciclato per W
    __shared__ DATA block_h2h[blockSide*blockSide];
    __shared__ DATA block_w[blockSide*blockSide];
    __shared__ DATA block_delta[blockSide*blockSide];

    temp_sum_delta_h2h[t_x+t_y*blockSide]=0.0f;

    for(int curr_patterns=0;curr_patterns<P;curr_patterns+=blockSide){
        pattern = (curr_patterns  + blockSide > P) ? (P-curr_patterns): blockSide ;

        int max_b_x = ((b_corner + blockSide) < B_right_limit) ? blockSide: (B_right_limit - b_corner);
        int max_a_x = ((a_corner + blockSide) < A_right_limit) ? blockSide: (A_right_limit - a_corner);

        DATA val = ((curr_patterns+ t_y) < P && max_b_x > t_x) ? delta[t_y*col2 + t_x + curr_patterns*col2]:0.0f;
        //DATA val_ai= ((curr_patterns+ t_y) < P && max_a_x > t_x) ? h2h[t_y*col1 + t_x +curr_patterns*col1]:0.0f;
        block_h2h[t_y*blockSide+t_x]= ((curr_patterns+ t_y) < P && max_a_x > t_x) ? h2h[t_y*col1 + t_x +curr_patterns*col1]:0.0f;
        
        for(int i=0 ;i<blockSide;i++){
           /*  if(t_y%2==0)//utile a spezzare il warp
                temp_shifted_mul[t_y][ t_x + i*blockSide ] =  val*warpGetVal(val_ai,i);
            else
                temp_shifted_mul[t_y][ t_x + i*blockSide ] =  val*warpGetVal(val_ai,16+i);
         */
           temp_shifted_mul[t_y][ t_x + i*blockSide ] =  val*block_h2h[t_y*blockSide+i];
           //temp_shifted_mul[t_y][ t_x + i*blockSide ] =  ( (curr_patterns+ t_y) < P && max_a_x > i) ? val*h2h[t_y*col1 + i +curr_patterns*col1]:0.0f;
        }    
        __syncthreads();
        
        if(t_y==0){
            for(int j=t_x,index=0; index<blockSide;j+=blockSide, index++ ){
                for(int i=0 ;i<pattern;i++)
                    temp_sum_delta_h2h[j] += temp_shifted_mul[i][j];
            }
        }
        __syncthreads();

    }
    if(t_y + a_corner < A_right_limit && t_x + b_corner < B_right_limit)
        thr_delta[t_x+t_y*col2] = temp_sum_delta_h2h[t_y*blockSide+ t_x];
    
}
/* THR_DEST has nupl[L]*nupl[l+1] element*/
__global__ void matrix_mul(DATA* H2H, DATA* W, DATA* DELTA, DATA* THR_DELTA, DATA* DEST_DELTA, int col1, int col2, int A_right_limit, int B_right_limit){
    int b_x = blockIdx.x*blockSide;
    int b_y = blockIdx.y*blockSide;

    matrix_array_block(H2H +b_y, W +b_x+b_y*col2, DELTA +b_x, THR_DELTA +b_x+b_y*col2, DEST_DELTA, col1, col2, A_right_limit, B_right_limit);
    __syncthreads();
}
void optimum_grid_x(dim3* grid,int max_block,int y_limit){
    
    int x = min((N+blockSide-1)/blockSide,max_block);
    int y=max_block/x;
    int prod=x*y;
    int new_prod;
    int new_x;

    for(new_x=x, new_prod=new_x*y ; new_prod != max_block && new_x > 1 && y < y_limit ;new_x--){
        y=max_block/new_x;
        new_prod=new_x*y;
    }

    if(new_prod>prod)
        x= new_x+1;

    grid->x = x;
    grid->y = y;

}
void backward(DATA *host_h2h, DATA* host_delta, DATA* host_thread_delta, DATA* d_h2h, DATA* d_w, DATA* d_delta, DATA* d_thread_delta, DATA* d_dest_delta, int col1, int col2){
    dim3 grid,block;
    optimum_grid_x(&grid,blockNum,col1/blockSide);
    block.x= blockSide;
    block.y= blockSide;
    printf("grid :%d %d\n",grid.y,grid.x);
    hipEvent_t start,stop;
    startTimer(&start,&stop);
    for(int sw_x=0; sw_x < col2; sw_x += grid.x*blockSide)
        for(int sw_y=0; sw_y < col1;sw_y += grid.y*blockSide) {
            matrix_mul<<< grid,block >>>(d_h2h +sw_y, d_w +sw_x+sw_y*col2, d_delta +sw_x, d_thread_delta+sw_x+sw_y*col2, d_dest_delta, col1, col2, min(col1-sw_y,grid.y*blockSide) ,min(col2-sw_x,grid.x*blockSide));
            //printf("grid :%d %d----limits:%d %d\n",sw_y,sw_x,min(col1-sw_y,grid.y*blockSide),min(col2-sw_x,grid.x*blockSide));
        }
    stopAndPrint(&start,&stop);

}

/*Check device*/
BOOL matsAreEquals(DATA *A, DATA *B, int rows, int cols) {

	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) { // the first column is for adapting the data
			float err = fabs(A[i*cols + j] - B[i*cols + j]);
			//printf("Error in i=%d,j=%d: %f\n", i, j, err);
			if (err >= epsilon) { printf("row: %d, col: %d\n", i, j); return 0; }
		}
	}
	return 1;
}

void printMat(DATA *mat, int rows, int cols) {

	for (int i = 0; i < rows; i++) {
		printf("ROW %d : {", i);
		for (int j = 0; j < cols; j++) {
			printf("%f - ", mat[i*cols + j]);
		}
		printf("}");
		printf("\n\n");
	}
	printf("\n\n");
}

int main(){
    DATA *h2h, *w, *delta, *c_host,*dest_c, *new_delta, *delta_host;
    DATA *d_h2h, *d_w, *d_delta, *d_thread_delta, *d_dest_delta;

    h2h=(DATA *)malloc(P*M*sizeof(DATA));
    w=(DATA *)malloc(M*N*sizeof(DATA));
    delta=(DATA *)malloc(P*N*sizeof(DATA));
    new_delta=(DATA *)calloc(P*M,sizeof(DATA));
    delta_host=(DATA *)calloc(P*M,sizeof(DATA));
    c_host=(DATA *)calloc(M*N,sizeof(DATA));
    dest_c=(DATA *)calloc(M*N,sizeof(DATA));

    hipMalloc((void**)&d_h2h,P*M*sizeof(DATA));
    hipMalloc((void**)&d_w,M*N*sizeof(DATA));
    hipMalloc((void**)&d_delta,P*N*sizeof(DATA));
    hipMalloc((void**)&d_dest_delta,P*M*sizeof(DATA));
    hipMalloc((void**)&d_thread_delta,M*N*sizeof(DATA));
/* -------------------------------init  -------------------*/
    for(int row=0;row<P;row++){
        for(int cola=0;cola<M;cola++)
            h2h[row*M+cola]=(DATA)rand() / (DATA)RAND_MAX;//1.0f;
        for(int colb=0;colb<N;colb++)
            delta[row*N+colb]=(DATA)rand() / (DATA)RAND_MAX;      
    }
    for(int cola=0;cola<M;cola++)
        for(int colb=0;colb<N;colb++)
            w[cola*N+colb]=(DATA)rand() / (DATA)RAND_MAX;//1.0f;
/*  -------------------------------------   */
    hipMemcpy(d_h2h,h2h,P*M*sizeof(DATA),hipMemcpyHostToDevice);
    hipMemcpy(d_w,w,P*M*sizeof(DATA),hipMemcpyHostToDevice);
    hipMemcpy(d_delta,delta,P*N*sizeof(DATA),hipMemcpyHostToDevice);
    hipMemcpy(d_thread_delta,c_host,N*M*sizeof(DATA),hipMemcpyHostToDevice);

    backward(h2h, delta, dest_c, d_h2h, d_w, d_delta, d_thread_delta, d_dest_delta, M, N);
    
    for(int row=0;row<P;row++){
        for(int cola=0;cola<M;cola++){
            DATA temp= 0.0f;
            for(int colb=0;colb<N;colb++)
                temp+= delta[row*N+colb]*w[cola*N+colb];    
            delta_host[row*M+cola] = temp*h2h[row*M+cola]*(1-h2h[row*M+cola]);
        }
    }

    for(int row=0;row<P;row++){
        for(int colb=0;colb<N;colb++)
            for(int cola=0;cola<M;cola++)
                c_host[cola*N+colb]+= h2h[row*M+cola]* delta[row*N+colb];      
    }

    hipMemcpy(dest_c,d_thread_delta,M*N*sizeof(DATA),hipMemcpyDeviceToHost);

    //printMat(c_host,M,N);
    printf("------------------------------\n");
    //printMat(dest_c,M,N);
    printf("------------------------------\n");
    matsAreEquals(dest_c,c_host,M,N);
    

    free(h2h);
    free(w);
    free(delta);
    free(delta_host);
    free(c_host);
    free(dest_c);
    hipFree(d_h2h);
    hipFree(d_w);
    hipFree(d_delta);
    hipFree(d_thread_delta);
    hipFree(d_dest_delta);
    return 0;
}