
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>

#define min(a,b) (a) < (b) ? (a) : (b)
#define BOOL int
#define blockSide 16
#define blockNum 8
#define epsilon 1e-1
#define N 56
#define M 784
#define P 60000
#define DATA float
#define eta 0.05f
#define alpha 0.8f
#define NSTREAMS 3
#define STREAMSIZE P / NSTREAMS

static void HandleCuda(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_CUDA( err ) (HandleCuda( err, __FILE__, __LINE__ ))

void startTimer(hipEvent_t *start, hipEvent_t *stop) {
	HANDLE_CUDA(hipEventCreate(start));
	HANDLE_CUDA(hipEventCreate(stop));
	HANDLE_CUDA(hipEventRecord(*start, 0));
}

void stopAndPrint(hipEvent_t *start, hipEvent_t *stop) {
	HANDLE_CUDA(hipEventRecord(*stop, 0));
	HANDLE_CUDA(hipEventSynchronize(*stop));
	float time = 0.0f;
	HANDLE_CUDA(hipEventElapsedTime(&time, *start, *stop));
	printf("Elapsed Time: %f milliseconds\n", time);
	HANDLE_CUDA(hipEventDestroy(*start));
	HANDLE_CUDA(hipEventDestroy(*stop));
}


/* la matrice di destinazione è width_h2h x width_delta     */
/* h2h_corner,delta_corner  sono in previsione di una "sliding grid" */
__device__ void MMMulDevPartialBack(DATA* h2h, DATA* w, DATA* delta, DATA* thr_delta_W, DATA* dest_delta, DATA* delta_weight_dest, DATA* delta_bias_dest, int width_h2h, int width_delta, int h2h_right_limit, int delta_right_limit, BOOL enable_bias){
    int t_x = threadIdx.x;
    int t_y = threadIdx.y;
    /* int idx = t_x + blockIdx.x*blockSide ; 
    int idy = t_y + blockIdx.y*blockSide ;  */
    int pattern;
    int h2h_corner = blockIdx.y*blockSide;
    int delta_corner = blockIdx.x*blockSide;

    __shared__ DATA temp_shifted_mul[blockSide][blockSide*blockSide];//può contenere diversi 0 nei casi sui bordi
    __shared__ DATA temp_sum_delta_h2h[blockSide*blockSide];//può essere riciclato per W
    __shared__ DATA block_h2h[blockSide*blockSide];
    __shared__ DATA block_w[blockSide*(blockSide+1)];//usefull for avoid bank conflict
    __shared__ DATA block_delta[blockSide*blockSide];
    __shared__ DATA bias_to_update[blockSide*blockSide];

    int max_b_x = ((delta_corner + blockSide) < delta_right_limit) ? blockSide: (delta_right_limit - delta_corner);
    int max_a_x = ((h2h_corner + blockSide) < h2h_right_limit) ? blockSide: (h2h_right_limit - h2h_corner);

    temp_sum_delta_h2h[t_x+t_y*blockSide]=0.0f;
    block_w[t_x*blockSide+t_y] = (max_a_x > t_y && max_b_x > t_x) ? w[t_y*width_delta + t_x]:0.0f;

    if(enable_bias==1)
        bias_to_update[t_y*blockSide + t_x] = 0.0f;

    for(int curr_patterns=0;curr_patterns<STREAMSIZE;curr_patterns+=blockSide){
        pattern = (curr_patterns  + blockSide > STREAMSIZE) ? (STREAMSIZE-curr_patterns): blockSide ;

        block_h2h[t_y*blockSide+t_x]= ((curr_patterns+ t_y) < STREAMSIZE && max_a_x > t_x) ? h2h[t_y*width_h2h + t_x +curr_patterns*width_h2h]:0.0f;
        block_delta[t_y*blockSide+t_x] = ((curr_patterns+ t_y) < STREAMSIZE && max_b_x > t_x) ? delta[t_y*width_delta + t_x + curr_patterns*width_delta]:0.0f;
        __syncthreads();
        //DATA val = ((curr_patterns+ t_y) < P && max_b_x > t_x) ? delta[t_y*width_delta + t_x + curr_patterns*width_delta]:0.0f;
        DATA val = block_delta[t_y*blockSide+t_x];
        DATA temp=0.0f;
        
        for(int i=0 ;i<blockSide;i++){
            temp += block_delta[t_y*blockSide+i]*block_w[i*blockSide+t_x];//product delta*W by trd[ty][tx]
            temp_shifted_mul[t_y][ t_x + i*blockSide ] =  val*block_h2h[t_y*blockSide+i];
        }    

        __syncthreads();
        if( t_y < pattern)
            atomicAdd(&dest_delta[t_y*width_h2h+ curr_patterns*width_h2h + t_x], temp*block_h2h[t_y*blockSide+t_x]*(1-block_h2h[t_y*blockSide+t_x]));//product 
        if(t_y==0){
            for(int j=t_x,index=0; index<blockSide;j+=blockSide, index++ ){
                for(int i=0 ;i<pattern;i++){
                    temp_sum_delta_h2h[j] += eta*temp_shifted_mul[i][j];
                }
            }
        }
        __syncthreads();
        if(enable_bias==1)//solo i blocchi con blocky = 0
            bias_to_update[t_y*blockSide + t_x] += block_delta[t_y*blockSide + t_x];
    }
    if( (t_y + h2h_corner) < h2h_right_limit && (t_x + delta_corner) < delta_right_limit){
        thr_delta_W[t_x+t_y*width_delta] = temp_sum_delta_h2h[t_y*blockSide+ t_x];
        delta_weight_dest[t_x+t_y*width_delta] = temp_sum_delta_h2h[t_y*blockSide+ t_x];
    }
    if(enable_bias==1 &&  t_y==0 && (t_x + delta_corner) < delta_right_limit){
        DATA tempBias=0.0f;
        for(int i=0;i<blockSide;i++)
            tempBias+=bias_to_update[i*blockSide+t_x];
        delta_bias_dest[t_x] = eta*tempBias ;
    }
    //__syncthreads();
}

/* si può la riduzione finale di W sommando i delta calcolati e riaggiornare quindi il delta W con gli stessi .
 oppure si fa prima ma bisogna salvarlo a parte e non sovrascrivere subito la matrice di partenza (problemi di concorrenza con altri stream) 
 la matrice di desinazione avrà streams*L*(L+1) elementi . si effettua la riduzione sullo stream principale per salvarla su quella giusta
 */


/* THR_DEST has nupl[L]*nupl[l+1] element*/



__device__ void MMMulReductionBlock(DATA* W, DATA* BIAS, DATA* DELTA_WEIGHT, DATA* DELTA_BIAS, DATA* DELTA_WEIGHT_DEST, DATA* DELTA_BIAS_DEST, int offset_weight, int offset_bias, int width_h2h, int width_delta,  int Y_right_limit, int X_right_limit, BOOL enable_bias){
    
    int t_x = threadIdx.x;
    int t_y = threadIdx.y;
    int b_x = blockIdx.x*blockSide;
    int b_y = blockIdx.y*blockSide;
    int offset_block_w = b_x+b_y*width_delta;
    int offset_block_bias = b_x;
 
    if( (b_x + t_x) < X_right_limit && (b_y+ t_y) < Y_right_limit){
       
        DATA dw_loc = alpha*DELTA_WEIGHT[t_x+t_y*width_delta];
        DATA dbias_loc;
                
        for(int i=0 ;i<NSTREAMS; i++){
            dw_loc += DELTA_WEIGHT_DEST[i*width_delta*width_h2h+ offset_weight + offset_block_w + t_x + t_y*width_delta];//offset della griglia + offset blocco + offset thr
            if(enable_bias==1 && t_y==0){
                dbias_loc += DELTA_BIAS_DEST[i*width_delta+ offset_bias + offset_block_bias + t_x];
                //printf("enable_bias [%d][%d]< %d >-- local %f, stream %d, get by stream %f\n",b_y+t_y,b_x+t_x,enable_bias, dbias_loc, i,DELTA_BIAS_DEST[i*width_delta+ offset_bias + offset_block_bias + t_x]);
            }
        }
        if(enable_bias==1 && t_y==0){
            //printf("enable_bias [%d][%d] -- DELTA BIAS:%f    local %f,  %f\n",b_y+t_y,b_x+t_x, DELTA_BIAS[t_x]+dbias_loc ,dbias_loc, alpha*DELTA_BIAS[t_x]);
            BIAS[t_x] += dbias_loc + alpha*DELTA_BIAS[t_x];
            DELTA_BIAS[t_x] = dbias_loc + alpha*DELTA_BIAS[t_x];
        }
        W[t_x + t_y*width_delta] += dw_loc;
        DELTA_WEIGHT[t_x + t_y*width_delta] = dw_loc;
        //printf("enable_bias [%d][%d] -- DELTA WEIGHT:%f    local %f  %d-%d\n",b_y+t_y,b_x+t_x, DELTA_WEIGHT[t_x + t_y*width_delta],dw_loc,Y_right_limit,X_right_limit);
    }
}

__global__ void MMMulReduction(DATA* W, DATA* BIAS, DATA* DELTA_WEIGHT, DATA* DELTA_BIAS, DATA* DELTA_WEIGHT_DEST, DATA* DELTA_BIAS_DEST, int offset_weight, int offset_bias, int width_h2h, int width_delta,  int Y_right_limit, int X_right_limit, BOOL enable_bias){
    int b_x = blockIdx.x*blockSide;
    int b_y = blockIdx.y*blockSide;
    //enable bias vale 1 se la griglia si è spostata lungo la x . Gli unici blocchi che calcoleranno il delta bias sono quelli con blockIdy = 0
    if(b_x < X_right_limit && b_y <Y_right_limit)
        MMMulReductionBlock(W+b_x+b_y*width_delta, BIAS+ b_x, DELTA_WEIGHT+b_x+b_y*width_delta, DELTA_BIAS+ b_x, DELTA_WEIGHT_DEST, DELTA_BIAS_DEST, offset_weight, offset_bias, width_h2h, width_delta,  Y_right_limit, X_right_limit, enable_bias*(1-blockIdx.y));
    //__syncthreads();
}
__global__ void MMMulDevBack(DATA* H2H, DATA* W, DATA* DELTA, DATA* THR_DELTA_W_H2H, DATA* DEST_DELTA,DATA* DELTA_WEIGHT_DEST, DATA* DELTA_BIAS_DEST, int width_h2h, int width_delta, int h2h_right_limit, int delta_right_limit, BOOL enable_bias){
    int b_x = blockIdx.x*blockSide;
    int b_y = blockIdx.y*blockSide;
    //enable bias vale 1 se la griglia si è spostata lungo la x . Gli unici blocchi che calcoleranno il delta bias sono quelli con blockIdy = 0
    if(b_x < delta_right_limit && b_y <h2h_right_limit)
        MMMulDevPartialBack(H2H +b_y, W +b_x+b_y*width_delta, DELTA +b_x, THR_DELTA_W_H2H +b_x+b_y*width_delta, DEST_DELTA+b_y, DELTA_WEIGHT_DEST +b_x+b_y*width_delta, DELTA_BIAS_DEST +b_x, width_h2h, width_delta, h2h_right_limit, delta_right_limit, enable_bias*(1-blockIdx.y));
    //__syncthreads();
}

void optimum_grid_x(dim3* grid,int max_block,int y_limit, int width_delta){
    
    int x = min((width_delta+blockSide-1)/blockSide,max_block);
    int y=max_block/x;
    int prod=x*y;
    int new_prod;
    int new_x;

    for(new_x=x, new_prod=new_x*y ; new_prod != max_block && new_x > 1 && y < y_limit ;new_x--){
        y=max_block/new_x;
        new_prod=new_x*y;
    }

    if(new_prod>prod)
        x= new_x+1;

    grid->x = x;
    grid->y = y;
}                                                                    
void backward(DATA *host_h2h, DATA* host_delta, DATA* host_thread_delta, DATA* d_h2h, DATA* d_w, DATA* d_bias, DATA* d_delta_weight, DATA* d_delta_bias, DATA* d_delta, DATA* d_thread_delta, DATA* d_dest_delta, DATA* d_delta_weight_dest, DATA* d_delta_bias_dest, int width_h2h, int width_delta, hipStream_t* streams){
    dim3 grid,block;
    optimum_grid_x(&grid,blockNum,width_h2h/blockSide,width_delta);
    block.x= blockSide;
    block.y= blockSide;
    printf("grid :%d %d\n",grid.y,grid.x);
    //cudaEvent_t start,stop;
    //startTimer(&start,&stop);
    for(int sw_x=0; sw_x < width_delta; sw_x += grid.x*blockSide){
        for(int sw_y=0; sw_y < width_h2h;sw_y += grid.y*blockSide) {
            for(int str=0;str<NSTREAMS;str++){
                MMMulDevBack<<< grid,block,0,streams[str]>>>(d_h2h +sw_y+str*STREAMSIZE*width_h2h, d_w +sw_x+sw_y*width_delta, d_delta +sw_x +str*STREAMSIZE*width_delta, d_thread_delta+sw_x+sw_y*width_delta, d_dest_delta + sw_y +str*STREAMSIZE*width_h2h, d_delta_weight_dest + str*width_h2h*width_delta +sw_x+sw_y*width_delta, d_delta_bias_dest+ str*width_delta +sw_x, width_h2h, width_delta, min(width_h2h-sw_y,grid.y*blockSide) ,min(width_delta-sw_x,grid.x*blockSide),(1-sw_y));
            }
        }
    }
    //cudaDeviceSynchronize();
    for(int sw_x=0; sw_x < width_delta; sw_x += grid.x*blockSide){
        for(int sw_y=0; sw_y < width_h2h;sw_y += grid.y*blockSide) {
            MMMulReduction<<<grid,block>>>(d_w +sw_x+sw_y*width_delta, d_bias+ sw_x, d_delta_weight +sw_x+sw_y*width_delta, d_delta_bias +sw_x , d_delta_weight_dest, d_delta_bias_dest, sw_x+sw_y*width_delta, sw_x,  width_h2h, width_delta, min(width_h2h-sw_y,grid.y*blockSide) ,min(width_delta-sw_x,grid.x*blockSide),(1-sw_y));
        }
    }
    //stopAndPrint(&start,&stop);
}

/*Check device*/
BOOL matsAreEquals(DATA *A, DATA *B, int rows, int cols) {

	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) { // the first column is for adapting the data
			float err = fabs(A[i*cols + j] - B[i*cols + j]);
			//printf("Error in i=%d,j=%d: %f\n", i, j, err);
			if (err >= epsilon) { printf("row: %d, col: %d----%f,%f\n", i, j,A[i*cols + j],B[i*cols + j]); return 0; }
		}
	}
	return 1;
}

void printMat(DATA *mat, int rows, int cols) {

	for (int i = 0; i < rows; i++) {
		printf("ROW %d : {", i);
		for (int j = 0; j < cols; j++) {
			printf("%f - ", mat[i*cols + j]);
		}
		printf("}");
		printf("\n\n");
	}
	printf("\n\n");
}

int main(){
    DATA *h2h, *w, *bias, *delta, *c_host, *dest_c, *new_delta, *delta_host, *delta_weight, *new_delta_weight, *delta_bias, *new_delta_bias;
    DATA *d_h2h, *d_w, *d_bias,*d_delta, *d_thread_delta, *d_dest_delta, *d_delta_weight, *d_delta_bias, *d_delta_weight_dest, *d_delta_bias_dest;
	
    h2h=(DATA *)malloc(P*M*sizeof(DATA));
    w=(DATA *)malloc(M*N*sizeof(DATA));
    bias=(DATA *)malloc(M*sizeof(DATA));
    delta=(DATA *)malloc(P*N*sizeof(DATA));//delta h2h
    new_delta=(DATA *)calloc(P*M,sizeof(DATA));
    delta_host=(DATA *)calloc(P*M,sizeof(DATA));
    c_host=(DATA *)calloc(M*N,sizeof(DATA));
    dest_c=(DATA *)calloc(M*N,sizeof(DATA));
    new_delta_weight=(DATA *)calloc(M*N,sizeof(DATA));
    delta_weight=(DATA *)calloc(M*N,sizeof(DATA));
    new_delta_bias=(DATA *)calloc(N,sizeof(DATA));
    delta_bias=(DATA *)calloc(N,sizeof(DATA));

    hipMalloc((void**)&d_h2h,P*M*sizeof(DATA));
    hipMalloc((void**)&d_w,M*N*sizeof(DATA));
    hipMalloc((void**)&d_bias,N*sizeof(DATA));
    hipMalloc((void**)&d_delta,P*N*sizeof(DATA));
    hipMalloc((void**)&d_dest_delta,P*M*sizeof(DATA));
    hipMalloc((void**)&d_thread_delta,M*N*sizeof(DATA));
    hipMalloc((void**)&d_delta_weight,M*N*sizeof(DATA));
    hipMalloc((void**)&d_delta_bias,N*sizeof(DATA));
    hipMalloc((void**)&d_delta_weight_dest,NSTREAMS*M*N*sizeof(DATA));
    hipMalloc((void**)&d_delta_bias_dest,NSTREAMS*N*sizeof(DATA));
    

    hipStream_t streams[NSTREAMS];
    /*++++-----------Streams creation------------++++*/
    for (int i = 0; i < NSTREAMS; i++) {
        HANDLE_CUDA(hipStreamCreate(&streams[i]));
    }
    /*---------------end--streams creation-----------*/
/* -------------------------------init  -------------------*/
    for(int row=0;row<P;row++){
        for(int cola=0;cola<M;cola++)
            h2h[row*M+cola]=(DATA)rand() / (DATA)RAND_MAX;
        for(int colb=0;colb<N;colb++)
            delta[row*N+colb]=(DATA)rand() / (DATA)RAND_MAX;      
    }
    for(int colb=0;colb<N;colb++){
        bias[colb]=(DATA)rand() / (DATA)RAND_MAX;
        delta_bias[colb]=(DATA)rand() / (DATA)RAND_MAX;
        for(int cola=0;cola<M;cola++){
            w[cola*N+colb]=(DATA)rand() / (DATA)RAND_MAX;
            delta_weight[cola*N+colb]=(DATA)rand() / (DATA)RAND_MAX;
        }
    }
/*  -------------------------------------   */
    hipMemcpy(d_h2h,h2h,P*M*sizeof(DATA),hipMemcpyHostToDevice);
    hipMemcpy(d_w,w,M*N*sizeof(DATA),hipMemcpyHostToDevice);
    hipMemcpy(d_bias,bias,N*sizeof(DATA),hipMemcpyHostToDevice);
    hipMemcpy(d_delta,delta,P*N*sizeof(DATA),hipMemcpyHostToDevice);
    hipMemcpy(d_dest_delta,new_delta,P*M*sizeof(DATA),hipMemcpyHostToDevice);
    hipMemcpy(d_thread_delta,c_host,M*N*sizeof(DATA),hipMemcpyHostToDevice);//parte di delta_weight nuovo
    hipMemcpy(d_delta_weight,delta_weight,M*N*sizeof(DATA),hipMemcpyHostToDevice);
    hipMemcpy(d_delta_bias,delta_bias,N*sizeof(DATA),hipMemcpyHostToDevice);
    //cudaMemcpy(d_delta_weight_dest,new_delta_weight,M*N*sizeof(DATA),cudaMemcpyHostToDevice);
    //cudaMemcpy(d_delta_bias_dest,new_delta_bias,N*sizeof(DATA),cudaMemcpyHostToDevice);

    backward(h2h, delta, dest_c, d_h2h, d_w, d_bias, d_delta_weight, d_delta_bias, d_delta, d_thread_delta, d_dest_delta, d_delta_weight_dest, d_delta_bias_dest, M, N,streams);
    for(int row=0;row<P;row++){
        for(int cola=0;cola<M;cola++){
            DATA temp= 0.0f;
            for(int colb=0;colb<N;colb++)
                temp+= delta[row*N+colb]*w[cola*N+colb];    
            delta_host[row*M+cola] = temp*h2h[row*M+cola]*(1.0f-h2h[row*M+cola]);
        }
    }
    for(int colb=0;colb<N;colb++){
        new_delta_bias[colb] = alpha*delta_bias[colb];
        for(int cola=0;cola<M;cola++)
            c_host[cola*N+colb] = alpha*delta_weight[cola*N+ colb];
    }
    for(int row=0;row<P;row++){
        for(int colb=0;colb<N;colb++){
            new_delta_bias[colb] += eta*delta[row*N+colb] ;
            for(int cola=0;cola<M;cola++)
                c_host[cola*N+colb]+= eta*h2h[row*M+cola]*delta[row*N+colb];
        }
    }

    hipMemcpy(dest_c,d_thread_delta, M*N*sizeof(DATA),hipMemcpyDeviceToHost);
    hipMemcpy(new_delta,d_dest_delta, P*M*sizeof(DATA),hipMemcpyDeviceToHost);
    hipMemcpy(delta_bias,d_delta_bias, N*sizeof(DATA),hipMemcpyDeviceToHost);
    hipMemcpy(delta_weight,d_delta_weight, M*N*sizeof(DATA),hipMemcpyDeviceToHost);


    printf(" delta W-h2h : \n");
    matsAreEquals(delta_weight,c_host,M,N);
    printf("------------------------------\n");
    //printMat(c_host,M,N);
    printf("------------------------------\n");
    //printMat(delta_weight,M,N);
    
    printf(" \ndelta h2h : \n");
    matsAreEquals(new_delta,delta_host,P,M);
    printf("------------------------------\n");
    //printMat(delta_host,20,M);
    printf("------------------------------\n");
    //printMat(new_delta,P,M);
  
    printf(" \ndelta bias : \n");
    matsAreEquals(new_delta_bias,delta_bias,1,N);
    /* printf("------------------------------\n");
    printMat(new_delta_bias,1,N);
    printf("------------------------------\n");
    printMat(delta_bias,1,N); */
 
    
        

    //printMat(new_delta,P,M);

    free(h2h);
    free(w);
    free(delta);
    free(delta_host);
    free(c_host);
    free(dest_c);
    hipFree(d_h2h);
    hipFree(d_w);
    hipFree(d_delta);
    hipFree(d_thread_delta);
    hipFree(d_dest_delta);
    return 0;
}